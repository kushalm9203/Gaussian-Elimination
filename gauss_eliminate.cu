#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>

// includes, kernels
#include "gauss_eliminate_kernel.cu"

#define MIN_NUMBER 2
#define MAX_NUMBER 50

extern "C" int compute_gold(float*, const float*, unsigned int);
Matrix allocate_matrix_on_gpu(const Matrix M);
Matrix allocate_matrix(int num_rows, int num_columns, int init);
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost);
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice);
void gauss_eliminate_on_device(const Matrix M, Matrix P);
void gauss_eliminate_on_device_optimized(const Matrix M, Matrix P);
int perform_simple_check(const Matrix M);
void print_matrix(const Matrix M);
void write_matrix_to_file(const Matrix M);
float get_random_number(int, int);
void checkCUDAError(const char *msg);
int checkResults(float *U, float *gpu_result, int num_elements, float threshold);


int 
main(int argc, char** argv) 
{
    // Matrices for the program
	Matrix  A; // The NxN input matrix
	Matrix  U; // The upper triangular matrix 
	struct timeval start, stop;
	// Initialize the random number generator with a seed value 
	srand(time(NULL));
	
	// Check command line arguments
	if(argc > 1){
		printf("Error. This program accepts no arguments. \n");
		exit(0);
	}		
	 
	// Allocate and initialize the matrices
	A  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 1);
	U  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0); 

	// Perform Gaussian elimination on the CPU 
	Matrix reference = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0);
	gettimeofday(&start, NULL);
	int status = compute_gold(U.elements, A.elements, A.num_rows);
	gettimeofday(&stop, NULL);
	printf("Execution time gold = %fs. \n", (float)(stop.tv_sec - start.tv_sec +\\
                (stop.tv_usec - start.tv_usec)/(float)1000000));
	if(status == 0){
		printf("Failed to convert given matrix to upper triangular. Try again. Exiting. \n");
		exit(0);
	}
	status = perform_simple_check(U); // Check that the principal diagonal elements are 1 
	if(status == 0){
		printf("The upper triangular matrix is incorrect. Exiting. \n");
		exit(0); 
	}
	printf("Gaussian elimination on the CPU was successful. \n");

	// Perform the vector-matrix multiplication on the GPU. Return the result in U
		gauss_eliminate_on_device(A,U);
		int num_elements = MATRIX_SIZE*MATRIX_SIZE;

		int res = checkResults(U.elements, U.elements, num_elements, 0.001f);
		printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

		gauss_eliminate_on_device_optimized(A, U);
    	// check if the device result is equivalent to the expected solution
	     res = checkResults(U.elements, U.elements, num_elements, 0.001f);
    printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

	// Free host matrices
	free(A.elements); A.elements = NULL;
	free(U.elements); U.elements = NULL;
	free(U.elements); U.elements = NULL;

	return 0;
}

void gauss_eliminate_on_device(const Matrix A, Matrix U)
{
		struct timeval start,stop;

	Matrix gpu_u = allocate_matrix_on_gpu( U );

	//Copy matrices to gpu, copy A right into U
	copy_matrix_to_device( gpu_u, A );
	
	int num_blocks = 1;
	
	int threads_per_block = 512;
	
	int ops_per_thread = MATRIX_SIZE / (threads_per_block*num_blocks);
	
	printf("== GPU (Slow) ==\n");
	printf("	Threads per block: %d\n",threads_per_block);
	printf("	Number of blocks: %d\n",num_blocks);
	printf("	Operations per thread: %d\n",ops_per_thread);
	
	dim3 thread_block(threads_per_block, 1, 1);
	dim3 grid(num_blocks,1);
	
	gettimeofday(&start, NULL);
	
	// Launch the kernel <<<grid, thread_block>>>
	gauss_eliminate_kernel<<<grid, thread_block>>>(gpu_u.elements,ops_per_thread);
	
	//Sync at end and check for errors
	hipDeviceSynchronize();
	checkCUDAError("FAST KERNEL FAILURE");
	gettimeofday(&stop, NULL);
	printf("Execution time GPU = %fs. \n", (float)(stop.tv_sec - start.tv_sec +\\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

		//Copy data back
	copy_matrix_from_device(U, gpu_u);
	
	//Free memory on device
	hipFree(gpu_u.elements);
	}

void 
gauss_eliminate_on_device_optimized(const Matrix A, Matrix U){
printf("== GPU (Fast) ==\n");
	Matrix gpu_u = allocate_matrix_on_gpu( U );

	//Copy matrices to gpu, copy A right into U
	copy_matrix_to_device( gpu_u, A );

	//Each thread within a block will take some j iterations
	int threads_per_block = 256; 
	struct timeval start,stop;
	int stride = threads_per_block;
	printf("	Threads per block / stride: %d\n",threads_per_block);

	
	
	int k;
	gettimeofday(&start, NULL);

	for(k = 0; k < MATRIX_SIZE; k++)
	{
		int isize = (MATRIX_SIZE-1) - (k+1) + 1;
		int num_blocks = isize;
		if(num_blocks <= 0)
		{
			num_blocks = 1;
		}
		
		dim3 thread_block(threads_per_block, 1, 1);
		dim3 grid(num_blocks,1);
	
		gauss_eliminate_kernel_optimized_div<<<grid, thread_block>>>(
			gpu_u.elements,
			k,
			stride);
		gauss_eliminate_kernel_optimized<<<grid, thread_block>>>(
			gpu_u.elements,
			k,
			stride);
			hipDeviceSynchronize();
			
		checkCUDAError("FAST KERNEL FAILURE");
	}
	gettimeofday(&stop, NULL);
	printf("Execution time GPU = %fs. \n", (float)(stop.tv_sec - start.tv_sec +\\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

	
	//Sync at end
	hipDeviceSynchronize();

	//Copy data back
	copy_matrix_from_device(U, gpu_u);
	
	//Free memory on device
	hipFree(gpu_u.elements);
	
	
	int i, j;
	for(i = 0; i < MATRIX_SIZE; i++)
			  for(j = 0; j < i; j++)
						 U.elements[i * MATRIX_SIZE + j] = 0.0;



}

// Allocate a device matrix of same size as M.
Matrix 
allocate_matrix_on_gpu(const Matrix M){
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix 
allocate_matrix(int num_rows, int num_columns, int init){
    	Matrix M;
    	M.num_columns = M.pitch = num_columns;
    	M.num_rows = num_rows;
    	int size = M.num_rows * M.num_columns;
		
	M.elements = (float*) malloc(size*sizeof(float));
	for(unsigned int i = 0; i < size; i++){
		if(init == 0) M.elements[i] = 0; 
		else
            M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void 
copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void 
copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice){
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

// Prints the matrix out to screen
void 
print_matrix(const Matrix M){
	for(unsigned int i = 0; i < M.num_rows; i++){
		for(unsigned int j = 0; j < M.num_columns; j++)
			printf("%f ", M.elements[i*M.num_rows + j]);
		printf("\n");
	} 
	printf("\n");
}

// Returns a random floating-point number between the specified min and max values 
float 
get_random_number(int min, int max){
	return (float)floor((double)(min + (max - min + 1)*((float)rand()/(float)RAND_MAX)));
}

// Performs a simple check on the upper triangular matrix. Checks to see if the principal diagonal elements are 1
int 
perform_simple_check(const Matrix M){
	for(unsigned int i = 0; i < M.num_rows; i++)
        if((fabs(M.elements[M.num_rows*i + i] - 1.0)) > 0.001) return 0;
	
    return 1;
} 

// Writes the matrix to a file 
void 
write_matrix_to_file(const Matrix M){
	FILE *fp;
	fp = fopen("matrix.txt", "wt");
	for(unsigned int i = 0; i < M.num_rows; i++){
        for(unsigned int j = 0; j < M.num_columns; j++)
            fprintf(fp, "%f", M.elements[i*M.num_rows + j]);
        }
    fclose(fp);
}

void 
checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}						 
}

int 
checkResults(float *U, float *gpu_result, int num_elements, float threshold)
{
    int checkMark = 1;
    float epsilon = 0.0;
    
    for(int i = 0; i < num_elements; i++)
        if(fabsf((U[i] - gpu_result[i])/U[i]) > threshold){
            checkMark = 0;
            break;
        }

    for(int i = 0; i < num_elements; i++)
        if(fabsf((U[i] - gpu_result[i])/U[i]) > epsilon){
            epsilon = fabsf((U[i] - gpu_result[i])/U[i]);
        }

    printf("Max epsilon = %f. \n", epsilon); 
    return checkMark;
}

